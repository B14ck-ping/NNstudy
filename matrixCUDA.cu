#include <stdio.h>
#include "matrixCUDA.h"
#include "kernels.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"


MatrixCUDA::MatrixCUDA(unsigned int a_rows, unsigned int a_columns) : rows(a_rows), columns(a_columns)
{
    hipMalloc( (void**)&matrix, rows*columns*sizeof(float) );
}

MatrixCUDA::MatrixCUDA(unsigned int a_rows, unsigned int a_columns, float value) : rows(a_rows), columns(a_columns)
{
    size_t size = rows*columns*sizeof(float);
    float *b_matrix = (float*)malloc(size);
    std::fill(b_matrix, b_matrix + (size / sizeof(float)), value);
    hipMalloc( (void**)&matrix, size);
    hipMemcpy( matrix, b_matrix, size, hipMemcpyHostToDevice );
    free(b_matrix);
}

MatrixCUDA::MatrixCUDA(MatrixCUDA &mtx)
{
    rows = mtx.get_rows();
    columns = mtx.get_columns();
    size_t size = rows*columns*sizeof(float);
    hipMalloc( (void**)&matrix, size);
    hipMemcpy( matrix, mtx.matrix, size, hipMemcpyDeviceToDevice ); 
}

MatrixCUDA::MatrixCUDA(const MatrixCUDA &mtx)
{
    rows = mtx.get_rows();
    columns = mtx.get_columns();
    size_t size = rows*columns*sizeof(float);
    hipMalloc( (void**)&matrix, size);
    hipMemcpy( matrix, mtx.matrix, size, hipMemcpyDeviceToDevice ); 
}

MatrixCUDA::MatrixCUDA(MatrixCUDA&& move_mtx)
{
    rows = move_mtx.get_rows();
    columns = move_mtx.get_columns();
    if (this->matrix != nullptr)
        hipFree(matrix);

    matrix = move_mtx.matrix;
    move_mtx.rows = 0;
    move_mtx.columns = 0;
    move_mtx.matrix = nullptr;
}

MatrixCUDA::MatrixCUDA(float *mtx_arr, unsigned int rows, unsigned int columns): rows(rows), columns(columns)
{
    size_t size = rows*columns*sizeof(float);
    hipMalloc( (void**)&matrix, size);
    hipMemcpy( matrix, mtx_arr, size, hipMemcpyHostToDevice ); 
}


MatrixCUDA::~MatrixCUDA()
{
    if(matrix != nullptr)
        hipFree(matrix);
}

MatrixCUDA MatrixCUDA::dot(const MatrixCUDA &mtx1, const MatrixCUDA &mtx2)
{
    if (mtx1.get_columns() != mtx2.get_rows())
        throw std::invalid_argument("Matrix multiplication error: columns of first matrix != rows of second matrix");

    unsigned common_size = mtx1.get_columns();
    unsigned output_rows = mtx1.get_rows();
    unsigned int output_columns = mtx2.get_columns();
    MatrixCUDA l_matrix(output_rows, output_columns);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 BlockDim((output_columns + TILE_SIZE - 1) / TILE_SIZE, (output_rows + TILE_SIZE - 1) / TILE_SIZE);
    dot_kernel<<<BlockDim, threadsPerBlock>>>(mtx1.matrix,mtx2.matrix, l_matrix.matrix, output_rows, common_size, output_columns);
    hipDeviceSynchronize();

    return l_matrix;
}

MatrixCUDA MatrixCUDA::operator* (float scalar)
{
    MatrixCUDA out_mtx(*this);
    out_mtx *= scalar;
    return out_mtx;
}

void MatrixCUDA::operator*= (float scalar)
{
    // запускаем add() kernel на GPU, передавая параметры
    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 BlockDim((columns + TILE_SIZE - 1) / TILE_SIZE, (rows + TILE_SIZE - 1) / TILE_SIZE);
    multiple_to_val<<<BlockDim, threadsPerBlock>>>(matrix, scalar, rows, columns);
    hipDeviceSynchronize();
}

MatrixCUDA MatrixCUDA::operator* (MatrixCUDA &m2)
{
    if (rows != m2.rows || columns != m2.columns) {
        std::cerr << "Matrix dimensions must match for addition!" << std::endl;
        return MatrixCUDA(0,0);
    }
    MatrixCUDA out_mtx(*this);
    out_mtx *= m2;
    return out_mtx;
}

void MatrixCUDA::operator*= (MatrixCUDA &m2)
{
    if (rows != m2.rows || columns != m2.columns) {
        std::cerr << "Matrix dimensions must match for addition!" << std::endl;
        return;
    }
    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 BlockDim((columns + TILE_SIZE - 1) / TILE_SIZE, (rows + TILE_SIZE - 1) / TILE_SIZE);
    multiple<<<BlockDim, threadsPerBlock>>>(matrix, m2.matrix, rows, columns);
    hipDeviceSynchronize();
}

MatrixCUDA MatrixCUDA::operator+ (MatrixCUDA &m2)
{
    if (rows != m2.rows || columns != m2.columns) {
        std::cerr << "Matrix dimensions must match for addition!" << std::endl;
        return MatrixCUDA(0,0);
    }
    MatrixCUDA out_mtx(*this);
    out_mtx += m2;
    return out_mtx;
}

void MatrixCUDA::operator+= (MatrixCUDA &m2)
{
    if (rows != m2.rows || columns != m2.columns) {
        std::cerr << "Matrix dimensions must match for addition!" << std::endl;
        return;
    }
    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 BlockDim((columns + TILE_SIZE - 1) / TILE_SIZE, (rows + TILE_SIZE - 1) / TILE_SIZE);
    increase<<<BlockDim, threadsPerBlock>>>(matrix, m2.matrix, rows, columns);
    hipDeviceSynchronize();
}

MatrixCUDA MatrixCUDA::operator- (MatrixCUDA& m2)
{
     if (rows != m2.rows || columns != m2.columns) {
        std::cerr << "Matrix dimensions must match for addition!" << std::endl;
        return MatrixCUDA(0,0);
    }
    MatrixCUDA out_mtx(*this);
    out_mtx -= m2;
    return out_mtx;
}

void MatrixCUDA::operator-= (MatrixCUDA &m2)
{
    if (rows != m2.rows || columns != m2.columns) {
        std::cerr << "Matrix dimensions must match for addition!" << std::endl;
        return;
    }
    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 BlockDim((columns + TILE_SIZE - 1) / TILE_SIZE, (rows + TILE_SIZE - 1) / TILE_SIZE);
    decrease<<<BlockDim, threadsPerBlock>>>(matrix, m2.matrix, rows, columns);
    hipDeviceSynchronize();
}

MatrixCUDA MatrixCUDA::operator= (MatrixCUDA &m2)
{
    this->rows = m2.rows;
    this->columns = m2.columns;

    if (this->matrix != nullptr)
        hipFree(matrix);

    size_t size = rows*columns*sizeof(float);
    hipMalloc( (void**)&matrix, size);
    hipMemcpy( matrix, m2.matrix, size, hipMemcpyDeviceToDevice ); 

    return *this;
}

MatrixCUDA MatrixCUDA::operator= (const MatrixCUDA &m2) 
{
    this->rows = m2.get_rows();
    this->columns = m2.get_columns();

    if (this->matrix != nullptr)
        hipFree(matrix);

    size_t size = rows*columns*sizeof(float);
    hipMalloc( (void**)&matrix, size);
    hipMemcpy( matrix, m2.matrix, size, hipMemcpyDeviceToDevice ); 

    return *this;
}

MatrixCUDA MatrixCUDA::operator= (MatrixCUDA&& move_mtx)
{
    this->rows = move_mtx.get_rows();
    this->columns = move_mtx.get_columns();

    if (this->matrix != nullptr)
        hipFree(matrix);

    this->matrix = move_mtx.matrix;
    move_mtx.rows = 0;
    move_mtx.columns = 0;
    move_mtx.matrix = nullptr;
    return *this;
}

float MatrixCUDA::getDeterminant() const
{
    return 0;
}

MatrixCUDA MatrixCUDA::getTranspose() const
{
    MatrixCUDA new_mtx(*this);
    new_mtx.transpose();
    return new_mtx;
}

void MatrixCUDA::transpose()
{
    float *t_matrix;

    size_t size = rows*columns*sizeof(float);
    hipMalloc( (void**)&t_matrix, size);
    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((columns + TILE_SIZE - 1) / TILE_SIZE, (rows + TILE_SIZE - 1) / TILE_SIZE);
    transpose_kernel<<<gridSize, blockSize>>>(matrix, t_matrix, columns, rows);
    hipDeviceSynchronize();

    hipFree(matrix);
    matrix = t_matrix;
    
    unsigned int temp = rows;
    rows = columns;
    columns = temp;
}


void MatrixCUDA::applySigmoid()
{
    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 BlockDim((columns + TILE_SIZE - 1) / TILE_SIZE, (rows + TILE_SIZE - 1) / TILE_SIZE);
    sygmoid<<<BlockDim, threadsPerBlock>>>(matrix, rows, columns);
    hipDeviceSynchronize();
}

float* MatrixCUDA::getHost_matrix() const
{
    size_t size = rows*columns*sizeof(float);
    float *l_buf = (float*)malloc(size);
    hipMemcpy( l_buf, matrix, size, hipMemcpyDeviceToHost); 

    return l_buf;
}

float* MatrixCUDA::getDeePCopyOnDevice() const
{
    float *l_buf = nullptr;
    hipMalloc( (void**)&l_buf, rows*columns*sizeof(float) );
    hipMemcpy( l_buf, matrix, rows*columns*sizeof(float), hipMemcpyDeviceToDevice ); 
    return l_buf;
}